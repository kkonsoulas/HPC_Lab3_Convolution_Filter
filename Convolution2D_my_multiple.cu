/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>



unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define ACCURACY 	0.5 

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}


//////////////////////////////////////////////////////////////////////////////////
//              CUDA CODE 
//////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, 
                       int imageW, int imageH, int filterR) {

  int k;
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += d_Src[y * imageW + d] * d_Filter[filterR - k];
        }     

        d_Dst[y * imageW + x] = sum;
      }
   
        
}


__global__ void convolutionColumnGPU(float *d_Dst, float *d_Src, float *d_Filter,
    			   int imageW, int imageH, int filterR) {

  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  int k;
 
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += d_Src[d * imageW + x] * d_Filter[filterR - k];
        }   
 
        d_Dst[y * imageW + x] = sum;
      }
   
}



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU,
    *d_Input,
    *d_Buffer,
    *d_Filter,
    *d_OutputGPU;


    int imageW;
    int imageH;
    unsigned int i;
    double accuracy = 0;

	printf("Enter filter radius : ");
	scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));

    //Check the mallocs
    if(h_Filter == NULL || h_Input == NULL || h_Buffer == NULL || h_OutputCPU == NULL || h_OutputGPU == NULL)
        return -1;

    hipMalloc((void**) &d_Filter,FILTER_LENGTH*sizeof(float));
    hipMalloc((void**) &d_OutputGPU,imageW * imageH *sizeof(float));
    hipMalloc((void**) &d_Buffer,imageW * imageH *sizeof(float));
    hipMalloc((void**) &d_Input,imageW * imageH *sizeof(float));

    
    //!Check also CUDA malloc!
    if(d_Filter == NULL || d_Input == NULL || d_Buffer == NULL || d_OutputGPU == NULL)
        return -1;



  
    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }


    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");
    clock_t start_t,stop_t;
    double total_t;

    start_t = clock(); 
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

    stop_t = clock();
    printf("CPU computation finished\n");
    
    total_t = (double)(stop_t - start_t) / CLOCKS_PER_SEC;
  




    
    dim3 grid_dim;
    dim3 block_dim;
    block_dim.x = imageW >= 32 ? 32 : imageW;
    block_dim.y = imageH >= 32 ? 32 : imageW;
    grid_dim.x = imageW >= 32 ? imageW/32 : 1;
    grid_dim.y = imageH >= 32 ? imageH/32 : 1;


    printf("GPU computation...\n");
    hipEvent_t start,stop;
    //timestamp GPU time
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice);

    convolutionRowGPU<<<grid_dim, block_dim>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    convolutionColumnGPU<<<grid_dim, block_dim>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius); 

    hipMemcpy(h_OutputGPU, d_OutputGPU, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost);
    
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    //timestamp GPU time
    printf("GPU computation finished\n");
    //print total GPU time.
    printf("Elapsed time in CPU: %lf (s)\n",total_t);
    printf("Elapsed time in GPU: %lf (s)\n",milliseconds/1000);

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
    for(int i=0; i < imageH * imageW; i++){
      if( (double) ABS(h_OutputCPU[i] - h_OutputGPU[i]) > accuracy){
        accuracy = (double) ABS((h_OutputCPU[i] - h_OutputGPU[i]));
        if(accuracy > ACCURACY){
          hipDeviceReset();
          fprintf(stderr,"Unexpected Diviation: %lf\nMax Permited Diviation: %lf\n",accuracy,ACCURACY);
          return(-1);  
        }
      }
    }
    fprintf(stderr,"Comparing results with accuracy %lf\n",accuracy);

    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}
