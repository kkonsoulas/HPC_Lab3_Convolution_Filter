/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define ACCURACY 	0.5 

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}


//////////////////////////////////////////////////////////////////////////////////
//              CUDA CODE 
//////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
  x = threadIdx.x;
  y = threadIdx.y;
  
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += d_Src[y * imageW + d] * d_Filter[filterR - k];
        }     

        d_Dst[y * imageW + x] = sum;
      }
   
        
}


__global__ void convolutionColumnGPU(float *d_Dst, float *d_Src, float *d_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  x = threadIdx.x;
  y = threadIdx.y;

 
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += d_Src[d * imageW + x] * d_Filter[filterR - k];
        }   
 
        d_Dst[y * imageW + x] = sum;
      }
   
}



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU,
    *d_Input,
    *d_Buffer,
    *d_Filter,
    *d_OutputGPU;


    int imageW;
    int imageH;
    unsigned int i;
    float accuracy = 0;

	printf("Enter filter radius : ");
	scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));

    //Check the mallocs
    if(h_Filter == NULL || h_Input == NULL || h_Buffer == NULL || h_OutputCPU == NULL || h_OutputGPU == NULL)
        return -1;

    hipMalloc((void**) &d_Filter,FILTER_LENGTH*sizeof(float));
    hipMalloc((void**) &d_OutputGPU,imageW * imageH *sizeof(float));
    hipMalloc((void**) &d_Buffer,imageW * imageH *sizeof(float));
    hipMalloc((void**) &d_Input,imageW * imageH *sizeof(float));

    
    //!Check also CUDA malloc!
    if(d_Filter == NULL || d_Input == NULL || d_Buffer == NULL || d_OutputGPU == NULL)
        return -1;



  
    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }


    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

    printf("CPU computation finished\n");

    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice);


    printf("GPU computation...\n");

    dim3 blocks(imageH,imageW);

    convolutionRowGPU<<<1, blocks>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    convolutionColumnGPU<<<1, blocks>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius); 

    printf("GPU computation finished\n");

    hipMemcpy(h_OutputGPU, d_OutputGPU, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost);
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
    for(int i=0; i < imageH * imageW; i++){
      if( (double) ABS(h_OutputCPU[i] - h_OutputGPU[i]) > accuracy){
        accuracy = (double) ABS((h_OutputCPU[i] - h_OutputGPU[i]));
        if(accuracy > ACCURACY){
          hipDeviceReset();
          fprintf(stderr,"Unexpected Diviation: %lf\nMax Permited Diviation: %lf\n",accuracy,ACCURACY);
          return(-1);  
        }
      }
    }
    fprintf(stderr,"Comparing results with accuracy %lf\n",accuracy);



    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}
